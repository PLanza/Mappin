#include "../../include/generate/grammar.hpp"
#include "../../include/gpu/actions.hpp"
#include "../../include/gpu/inet.hpp"
#include "../../include/gpu/kernel.cuh"
#include "../../include/gpu/network.hpp"
#include "../../include/gpu/queue.cuh"
#include "../../include/gpu/run.hpp"
#include "../../include/gpu/timing.cuh"

#include <cstdlib>
#include <iostream>
#include <memory>
#include <string>

void _checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(err) _checkCudaErrors(err, __FILE__, __LINE__)

void parse(std::unique_ptr<grammar::Grammar> grammar,
           std::string &input_string) {
  std::cout << "Parsing: " << input_string << std::endl;

  dim3 grid_dims(GRID_DIM_X, 1, 1);
  dim3 block_dims(BLOCK_DIM_X, 1, 1);

  initActions();

  copyConstantData();

  // Set up starting interaction network
  std::vector<grammar::Token> tokens = grammar->stringToTokens(input_string);
  HostINetwork host_network(grammar->getStackActions(), tokens);

  size_t interactions_size = host_network.getInteractions();
  size_t network_size = host_network.getNetworkSize();
  std::cout << "Network is made of " << network_size << " nodes." << std::endl;
  std::cout << sizeof(actions_map_h) << std::endl;

  Interaction *interactions =
      (Interaction *)malloc(interactions_size * sizeof(Interaction));

  // Allocate network
  NodeElement *network_d;
  checkCudaErrors(
      hipMalloc((void **)&network_d, sizeof(NodeElement) * network_size));
  host_network.initNetwork(network_d, interactions);

  // Initialize global queue such that the first set of interactions can be
  // immediately loaded by the threads
  InteractionQueue<MAX_INTERACTIONS_SIZE> *globalQueue_h =
      new InteractionQueue<MAX_INTERACTIONS_SIZE>(interactions,
                                                  interactions_size);
  InteractionQueue<MAX_INTERACTIONS_SIZE> *globalQueue_d;
  checkCudaErrors(hipMalloc((void **)&globalQueue_d,
                             sizeof(InteractionQueue<MAX_INTERACTIONS_SIZE>)));
  checkCudaErrors(hipMemcpy(globalQueue_d, globalQueue_h,
                             sizeof(InteractionQueue<MAX_INTERACTIONS_SIZE>),
                             hipMemcpyHostToDevice));

  hipDeviceSetLimit(hipLimitMallocHeapSize,
                     MAX_INTERACTIONS_SIZE * sizeof(Interaction) +
                         MAX_NETWORK_SIZE * sizeof(NodeElement));

  bool *global_done_d;
  checkCudaErrors(
      hipMalloc((void **)&global_done_d, sizeof(bool) * grid_dims.x));

  NodeElement *output_network_d;
  checkCudaErrors(hipMalloc((void **)&output_network_d,
                             sizeof(NodeElement) * network_size));

  struct hipFuncAttributes funcAttrib;
  checkCudaErrors(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(runINet)));
  printf("%s numRegs=%d\n", "runINet", funcAttrib.numRegs);

  // Timing
  Timing *timing = new Timing();
  timing->StartCounter();

  // while (true) {
  //   int32_t gq_count_h;
  //   checkCudaErrors(hipMemcpy(&gq_count_h, &globalQueue_d->count,
  //                              sizeof(int32_t), hipMemcpyDeviceToHost));
  //
  //   if (gq_count_h == 0)
  //     break;
  //
  //   uint32_t gq_head_h;
  //   checkCudaErrors(hipMemcpy(&gq_head_h, &globalQueue_d->head,
  //                              sizeof(uint32_t), hipMemcpyDeviceToHost));
  //
  //   // Clear the queue
  //   checkCudaErrors(hipMemcpy(&globalQueue_d->head, &globalQueue_d->tail,
  //                              sizeof(uint32_t), hipMemcpyDeviceToDevice));
  //   checkCudaErrors(hipMemset(&globalQueue_d->count, 0, sizeof(uint32_t)));
  //
  //   // Dynamically launch the kernel
  //   uint32_t grid_dimx = gq_count_h / BLOCK_DIM_X + 1;
  //   resolveINets<<<grid_dimx, block_dims>>>(globalQueue_d, gq_head_h,
  //                                           gq_count_h, network_d);
  //   hipDeviceSynchronize();
  // }

  // Invoke kernel
  runINet<<<grid_dims, block_dims>>>(globalQueue_d, global_done_d, network_d);
  hipDeviceSynchronize();

  std::cout << "Parsing took " << timing->GetCounter() << " ms" << std::endl;
  timing->StartCounter();

  copyNetwork<<<grid_dims, block_dims>>>(network_d + network_size - 5,
                                         output_network_d, globalQueue_d);
  std::cout << "Copying the network took " << timing->GetCounter() << " ms"
            << std::endl;

  uint64_t output_net_size;
  checkCudaErrors(hipMemcpy(&output_net_size, network_d + network_size - 5,
                             sizeof(uint64_t), hipMemcpyDeviceToHost));
  std::cout << "Output network has " << output_net_size << " NodeElements"
            << std::endl;

  NodeElement *output_network_h =
      (NodeElement *)malloc(sizeof(NodeElement) * network_size);
  checkCudaErrors(hipMemcpy(output_network_h, output_network_d,
                             sizeof(NodeElement) * output_net_size,
                             hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(globalQueue_d));
  checkCudaErrors(hipFree(network_d));

  std::cout << "Parsing results: " << std::endl;
  std::vector<grammar::ParseTree *> trees =
      grammar->getParses(output_network_h, output_network_d);
  for (grammar::ParseTree *tree : trees) {
    grammar->printParseTree(tree);
    std::cout << std::endl;
  }
}
