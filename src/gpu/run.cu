#include "../../include/generate/grammar.hpp"
#include "../../include/gpu/actions.hpp"
#include "../../include/gpu/inet.hpp"
#include "../../include/gpu/kernel.cuh"
#include "../../include/gpu/network.hpp"
#include "../../include/gpu/queue.cuh"
#include "../../include/gpu/run.hpp"
#include "../../include/gpu/timing.cuh"

#include <cstdlib>
#include <iostream>
#include <memory>
#include <string>

void _checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(err) _checkCudaErrors(err, __FILE__, __LINE__)

void runInteractionNet(InteractionQueue<MAX_INTERACTIONS_SIZE> *global_queue_d,
                       NodeElement *network_d) {

  uint32_t total_inters = 0;
  unsigned long long queue_head;
  int32_t queue_count;
  do {
    checkCudaErrors(hipMemcpy(&queue_head, &global_queue_d->head,
                               sizeof(unsigned long long),
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&queue_count, &global_queue_d->count,
                               sizeof(int32_t), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&global_queue_d->head, &global_queue_d->tail,
                               sizeof(unsigned long long),
                               hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemset(&global_queue_d->count, 0, sizeof(int32_t)));

    uint32_t grid_dim_x = queue_count / BLOCK_DIM_X + 1;
    total_inters += queue_count;

    reduceInteractions<<<grid_dim_x, BLOCK_DIM_X>>>(global_queue_d, network_d,
                                                    queue_count, queue_head);
    hipDeviceSynchronize();

    std::cout << "Total interactions so far: " << total_inters << "\n\n";

  } while (queue_count != 0);
}

void parse(std::unique_ptr<grammar::Grammar> grammar,
           std::string &input_string) {
  std::cout << "Parsing: " << input_string << std::endl;

  dim3 grid_dims(GRID_DIM_X, 1, 1);
  dim3 block_dims(BLOCK_DIM_X, 1, 1);

  initActions();

  copyConstantData();

  // Set up starting interaction network
  std::vector<grammar::Token> tokens = grammar->stringToTokens(input_string);
  HostINetwork host_network(grammar->getStackActions(), tokens);

  size_t interactions_size = host_network.getInteractions();
  size_t network_size = host_network.getNetworkSize();
  std::cout << "Network is made of " << network_size << " nodes." << std::endl;
  std::cout << sizeof(actions_map_h) << std::endl;

  Interaction *interactions =
      (Interaction *)malloc(interactions_size * sizeof(Interaction));

  // Allocate network
  NodeElement *network_d;
  checkCudaErrors(
      hipMalloc((void **)&network_d, sizeof(NodeElement) * network_size));
  host_network.initNetwork(network_d, interactions);

  // Initialize global queue such that the first set of interactions can be
  // immediately loaded by the threads
  InteractionQueue<MAX_INTERACTIONS_SIZE> *globalQueue_h =
      new InteractionQueue<MAX_INTERACTIONS_SIZE>(interactions,
                                                  interactions_size);
  InteractionQueue<MAX_INTERACTIONS_SIZE> *globalQueue_d;
  checkCudaErrors(hipMalloc((void **)&globalQueue_d,
                             sizeof(InteractionQueue<MAX_INTERACTIONS_SIZE>)));
  checkCudaErrors(hipMemcpy(globalQueue_d, globalQueue_h,
                             sizeof(InteractionQueue<MAX_INTERACTIONS_SIZE>),
                             hipMemcpyHostToDevice));

  hipDeviceSetLimit(hipLimitMallocHeapSize,
                     MAX_INTERACTIONS_SIZE * sizeof(Interaction) +
                         MAX_NETWORK_SIZE * sizeof(NodeElement));

  bool *global_done_d;
  checkCudaErrors(
      hipMalloc((void **)&global_done_d, sizeof(bool) * grid_dims.x));

  NodeElement *output_network_d;
  checkCudaErrors(hipMalloc((void **)&output_network_d,
                             sizeof(NodeElement) * network_size));

  struct hipFuncAttributes funcAttrib;
  checkCudaErrors(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(reduceInteractions)));
  printf("%s numRegs=%d\n", "runINet", funcAttrib.numRegs);

  // Timing
  Timing *timing = new Timing();
  timing->StartCounter();

  runInteractionNet(globalQueue_d, network_d);

  // Invoke kernel
  // runINet<<<grid_dims, block_dims>>>(globalQueue_d, global_done_d,
  // network_d); hipDeviceSynchronize();

  std::cout << "Parsing took " << timing->GetCounter() << " ms" << std::endl;
  timing->StartCounter();

  uint32_t grid_dim_x = tokens.size() / BLOCK_DIM_X + 1;

  copyNetwork<<<grid_dim_x, block_dims>>>(network_d + network_size - 5,
                                          output_network_d, globalQueue_d);
  std::cout << "Copying the network took " << timing->GetCounter() << " ms"
            << std::endl;

  uint64_t output_net_size;
  checkCudaErrors(hipMemcpy(&output_net_size, network_d + network_size - 5,
                             sizeof(uint64_t), hipMemcpyDeviceToHost));
  std::cout << "Output network has " << output_net_size << " NodeElements"
            << std::endl;

  NodeElement *output_network_h =
      (NodeElement *)malloc(sizeof(NodeElement) * network_size);
  checkCudaErrors(hipMemcpy(output_network_h, output_network_d,
                             sizeof(NodeElement) * output_net_size,
                             hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(globalQueue_d));
  checkCudaErrors(hipFree(network_d));

  std::cout << "Parsing results: " << std::endl;
  std::vector<grammar::ParseTree *> trees =
      grammar->getParses(output_network_h, output_network_d);
  for (grammar::ParseTree *tree : trees) {
    grammar->printParseTree(tree);
    std::cout << std::endl;
  }
}
