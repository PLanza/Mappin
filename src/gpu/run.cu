#include "../../include/generate/grammar.hpp"
#include "../../include/gpu/actions.hpp"
// #include "../../include/gpu/draw.hpp"
#include "../../include/gpu/inet.hpp"
#include "../../include/gpu/kernel.cuh"
#include "../../include/gpu/network.hpp"
#include "../../include/gpu/queue.cuh"
#include "../../include/gpu/run.hpp"
#include "../../include/gpu/timing.cuh"

#include <cstdint>
#include <cstdlib>
#include <iostream>
#include <memory>
#include <string>
#include <vector>

void _checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(err) _checkCudaErrors(err, __FILE__, __LINE__)

NodeElement *copyInteractionNet(
    NodeElement *output_node, NodeElement *src_network,
    NodeElement *dst_network, uint32_t *net_size,
    ParallelQueue<uint32_t, MAX_INTERACTIONS_SIZE> *copy_queue,
    ParallelQueue<Interaction, MAX_INTERACTIONS_SIZE> *inters_queue) {
  unsigned long long copy_queue_head = 0;
  uint32_t value = 1;
  // Copy the output node
  checkCudaErrors(hipMemcpy(dst_network + 1, output_node,
                             sizeof(NodeElement) * 6,
                             hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy(&(output_node + 1)->flags, &value,
                             sizeof(uint32_t), hipMemcpyHostToDevice));

  // Reset the copy queue
  checkCudaErrors(hipMemset(&copy_queue->head, 0, sizeof(unsigned long long)));
  checkCudaErrors(hipMemset(&copy_queue->tail, 0, sizeof(unsigned long long)));

  // Add the first node to the copy queue
  checkCudaErrors(hipMemcpy(&copy_queue->buffer, output_node + 4,
                             sizeof(NodeElement), hipMemcpyDeviceToDevice));

  value += 6;
  checkCudaErrors(
      hipMemcpy(net_size, &value, sizeof(uint32_t), hipMemcpyHostToDevice));

  int32_t queue_count = 1;
  while (queue_count != 0) {
    checkCudaErrors(hipMemcpy(&copy_queue_head, &copy_queue->head,
                               sizeof(unsigned long long),
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&copy_queue->head, &copy_queue->tail,
                               sizeof(unsigned long long),
                               hipMemcpyDeviceToDevice));

    checkCudaErrors(hipMemset(&copy_queue->count, 0, sizeof(int32_t)));
    uint32_t grid_dim_x = queue_count / BLOCK_DIM_X + 1;

    copyNetwork<<<grid_dim_x, BLOCK_DIM_X>>>(src_network, net_size, dst_network,
                                             copy_queue, queue_count,
                                             copy_queue_head);
    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(&queue_count, &copy_queue->count,
                               sizeof(int32_t), hipMemcpyDeviceToHost));
  }

  // Rewire the interactions
  uint32_t inters_count;
  checkCudaErrors(hipMemcpy(&inters_count, &inters_queue->count,
                             sizeof(int32_t), hipMemcpyDeviceToHost));
  uint32_t grid_dim_x = inters_count / BLOCK_DIM_X + 1;

  rewireInteractions<<<grid_dim_x, BLOCK_DIM_X>>>(src_network, inters_queue);

  return dst_network + 1;
}

uint32_t total_inters = 0;
NodeElement *runInteractionNet(
    ParallelQueue<Interaction, MAX_INTERACTIONS_SIZE> *inters_queue_d,
    ParallelQueue<uint32_t, MAX_INTERACTIONS_SIZE> *copy_queue_d,
    NodeElement *&network_d, NodeElement *&network_copy_d, uint32_t *net_size_d,
    HostINetwork &starting_net) {

  unsigned long long queue_head = 0;
  int32_t queue_count = starting_net.interactions.size();

  uint32_t net_size_h;
  NodeElement *output_node = network_d + starting_net.network.size() - 6;

  uint32_t iters = 0;

  while (queue_count != 0) {
    iters += 1;
    checkCudaErrors(hipMemcpy(&queue_head, &inters_queue_d->head,
                               sizeof(unsigned long long),
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&inters_queue_d->head, &inters_queue_d->tail,
                               sizeof(unsigned long long),
                               hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemset(&inters_queue_d->count, 0, sizeof(int32_t)));

    uint32_t grid_dim_x = queue_count / BLOCK_DIM_X + 1;
    total_inters += queue_count;

    reduceNetwork<<<grid_dim_x, BLOCK_DIM_X>>>(
        inters_queue_d, network_d, net_size_d, queue_count, queue_head);
    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(&net_size_h, net_size_d, sizeof(uint32_t),
                               hipMemcpyDeviceToHost));

    // Drawing
    // std::cout << "Size before collection: " << net_size_h << std::endl;
    //
    // output_node = copyInteractionNet(output_node, network_d, network_copy_d,
    //                                  net_size_d, copy_queue_d,
    //                                  inters_queue_d);
    // NodeElement *tmp = network_d;
    // network_d = network_copy_d;
    // network_copy_d = tmp;
    //
    // checkCudaErrors(hipMemcpy(&net_size_h, net_size_d, sizeof(uint32_t),
    //                            hipMemcpyDeviceToHost));
    // std::cout << "Size after collection: " << net_size_h << std::endl;
    // NodeElement *network_h =
    //     (NodeElement *)malloc(sizeof(NodeElement) * net_size_h);
    // checkCudaErrors(hipMemcpy(network_h, network_d,
    //                            sizeof(NodeElement) * net_size_h,
    //                            hipMemcpyDeviceToHost));
    // drawNetwork(network_h);
    // End Drawing

    checkCudaErrors(hipMemcpy(&queue_count, &inters_queue_d->count,
                               sizeof(int32_t), hipMemcpyDeviceToHost));

    // If nearing the limit network size copy collect it
    if (net_size_h + queue_count * 40 > MAX_NETWORK_SIZE) {
      Timing *timing = new Timing();
      timing->StartCounter();

      std::cout << "Copy collecting network" << std::endl;
      output_node =
          copyInteractionNet(output_node, network_d, network_copy_d, net_size_d,
                             copy_queue_d, inters_queue_d);
      NodeElement *tmp = network_d;
      network_d = network_copy_d;
      network_copy_d = tmp;
      std::cout << "Copying took " << timing->GetCounter() << " ms"
                << std::endl;
    }
    // std::cout << "Total interactions so far: " << total_inters << "\n\n";
  }
  std::cout << iters << " iterations" << std::endl;

  return output_node;
}

void parse(std::unique_ptr<grammar::Grammar> grammar,
           std::string &input_string) {
  std::cout << "Parsing: " << input_string << std::endl;

  struct hipFuncAttributes funcAttrib;
  checkCudaErrors(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(reduceNetwork)));
  printf("%s numRegs=%d\n", "reduceNetwork", funcAttrib.numRegs);

  initActions();

  copyConstantData();

  Timing *timing = new Timing();
  timing->StartCounter();

  // Set up starting interaction network
  std::vector<grammar::Token> tokens = grammar->stringToTokens(input_string);
  std::cout << tokens.size() << std::endl;
  HostINetwork host_network(grammar->getStackActions(), tokens);
  uint32_t net_size_h = host_network.network.size();

  std::cout << "Network is made of " << net_size_h << " node elements."
            << std::endl;
  std::cout << sizeof(actions_map_h) << std::endl;

  // Allocate network
  NodeElement *network_d;
  checkCudaErrors(
      hipMalloc((void **)&network_d, sizeof(NodeElement) * MAX_NETWORK_SIZE));
  checkCudaErrors(hipMemcpy(network_d, host_network.network.data(),
                             sizeof(NodeElement) * net_size_h,
                             hipMemcpyHostToDevice));

  NodeElement *network_copy_d;
  checkCudaErrors(hipMalloc((void **)&network_copy_d,
                             sizeof(NodeElement) * MAX_NETWORK_SIZE));
  uint32_t *net_size_d;
  checkCudaErrors(hipMalloc((void **)&net_size_d, sizeof(uint32_t)));
  checkCudaErrors(hipMemcpy(net_size_d, &net_size_h, sizeof(uint32_t),
                             hipMemcpyHostToDevice));

  // Initialize interaction queue
  ParallelQueue<Interaction, MAX_INTERACTIONS_SIZE> *inters_queue_d =
      newParallelQueue<Interaction, MAX_INTERACTIONS_SIZE>(
          host_network.interactions);

  // Initialize copy queue
  ParallelQueue<uint32_t, MAX_INTERACTIONS_SIZE> *copy_queue_d =
      newParallelQueue<uint32_t, MAX_INTERACTIONS_SIZE>(
          std::vector<uint32_t>());

  std::cout << "Setup took " << timing->GetCounter() << " ms" << std::endl;
  timing->StartCounter();

  NodeElement *output_node =
      runInteractionNet(inters_queue_d, copy_queue_d, network_d, network_copy_d,
                        net_size_d, host_network);

  std::cout << "Parsing took " << timing->GetCounter() << " ms" << std::endl;
  timing->StartCounter();

  copyInteractionNet(output_node, network_d, network_copy_d, net_size_d,
                     copy_queue_d, inters_queue_d);
  std::cout << "Copying the network took " << timing->GetCounter() << " ms"
            << std::endl;

  // Get network size
  uint32_t output_net_size;
  checkCudaErrors(hipMemcpy(&output_net_size, net_size_d, sizeof(uint32_t),
                             hipMemcpyDeviceToHost));
  std::cout << "Output network has " << output_net_size << " NodeElements"
            << std::endl;
  std::cout << "Total interactions: " << total_inters << "\n\n";

  // Copy output network
  NodeElement *output_network_h =
      (NodeElement *)malloc(sizeof(NodeElement) * net_size_h);
  checkCudaErrors(hipMemcpy(output_network_h, network_copy_d,
                             sizeof(NodeElement) * output_net_size,
                             hipMemcpyDeviceToHost));

  // Get parsing results
  std::cout << "Parsing results: " << std::endl;
  std::vector<grammar::ParseTree *> trees =
      grammar->getParses(output_network_h);
  for (grammar::ParseTree *tree : trees) {
    grammar->printParseTree(tree);
    std::cout << std::endl;
  }

  checkCudaErrors(hipFree(inters_queue_d));
  checkCudaErrors(hipFree(network_d));
  checkCudaErrors(hipFree(network_copy_d));
  checkCudaErrors(hipFree(net_size_d));
}
