#include "../../include/gpu/timing.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

struct PrivateTiming {
  hipEvent_t start;
  hipEvent_t stop;
};

// default constructor
Timing::Timing() { privateTiming = new PrivateTiming; }

// default destructor
Timing::~Timing() {}

void Timing::StartCounter() {
  hipEventCreate(&((*privateTiming).start));
  hipEventCreate(&((*privateTiming).stop));
  hipEventRecord((*privateTiming).start, 0);
}

void Timing::StartCounterFlags() {
  int eventflags = hipEventBlockingSync;

  hipEventCreateWithFlags(&((*privateTiming).start), eventflags);
  hipEventCreateWithFlags(&((*privateTiming).stop), eventflags);
  hipEventRecord((*privateTiming).start, 0);
}

// Gets the counter in ms
float Timing::GetCounter() {
  float time;
  hipEventRecord((*privateTiming).stop, 0);
  hipEventSynchronize((*privateTiming).stop);
  hipEventElapsedTime(&time, (*privateTiming).start, (*privateTiming).stop);
  return time;
}
