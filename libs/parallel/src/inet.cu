#include "hip/hip_runtime.h"
#include "../include/parallel/actions.cuh"
#include "../include/parallel/inet.cuh"
#include "../include/parallel/queue.cuh"
#include <cstdint>

const uint8_t NODE_ARITIES_H[NODE_KINDS] = {
    1, 0, 2, 2, 0, 2, 2, 3, 3, 0, 3, 3, 2, 2, 2, 1, 1, 1, 0, 1, 1, 2,
};

#define BLOCK_QUEUE_SIZE 4 * 256
#define MAX_NEW_NODES 3

__device__ inline uint32_t actMapIndex(uint8_t left, uint8_t right) {
  return (left * (2 * NODE_KINDS - left + 1) / 2 + right - left) * 2 *
         MAX_ACTIONS;
}

__global__ void runINet(NodeElement *network,
                        InteractionQueue<MAX_INTERACTIONS_SIZE> *globalQueue,
                        size_t inters_size, bool *global_done) {

  Interaction interact_buf[5];
  uint8_t buf_elems = 1;

  __shared__ uint8_t *block_done;

  __shared__ Interaction block_queue[BLOCK_QUEUE_SIZE]; // 16384
  __shared__ uint32_t count, head, tail;

  __shared__ int64_t global_queue_idx;

  // Copy initial interactions
  if (blockDim.x * blockIdx.x + threadIdx.x < inters_size)
    interact_buf[0] =
        globalQueue->buffer[blockDim.x * blockIdx.x + threadIdx.x];

  if (threadIdx.x == 0) {
    count = 0;
    head = 0;
    tail = 0;

    // might want to statically allocate this
    block_done = (uint8_t *)malloc(sizeof(uint8_t) * blockDim.x / 8);
    memset(block_done, 0, sizeof(uint8_t) * blockDim.x / 8);
  }
  __syncthreads();

  while (true) {
    // TODO: optimize this first section so that spinning is more efficient

    // Might need this to be atomic
    block_done[threadIdx.x / 8] |= ((uint8_t)buf_elems == 0)
                                   << (threadIdx.x % 8);

    // Set first bit to 0 if the queues are not empty
    if (threadIdx.x == 0)
      block_done[0] &= ((uint8_t)~0)
                       << (globalQueue->isEmpty() || head == tail);

    // If all threads in block are done
    if (__syncthreads_and(block_done[threadIdx.x / 8] == (uint8_t)~0u)) {
      global_done[blockIdx.x] = true;
      // If all blocks are done
      if (__syncthreads_and(global_done[threadIdx.x % gridDim.x]))
        // Might need to synchronize across grid
        break;

      continue;
    }

    // Attempt to dequeue block_queue if it's full
    if (count == BLOCK_QUEUE_SIZE) {
      if (threadIdx.x == 0)
        globalQueue->enqueue(&global_queue_idx, 3 * blockDim.x);
      __syncthreads();

      // If both the block queue and the global queue are full then spin
      if (global_queue_idx == -1)
        continue;

      // Otherwise copy data from block to global queue
      for (int i = 0; i < 3; i++) {
        globalQueue->buffer[global_queue_idx + i * blockDim.x + threadIdx.x] =
            block_queue[head + i * blockDim.x + threadIdx.x];
      }
      __syncthreads();
      if (threadIdx.x == 0) {
        head += 3 * blockDim.x;
        count -= 3 * blockDim.x;
        globalQueue->ackEnqueue(3 * blockDim.x);
      }
    }

    // Attempt to enqueue block_queue from global if nearing empty
    if (count < blockDim.x) {
      if (threadIdx.x == 0)
        globalQueue->enqueue(&global_queue_idx, blockDim.x - count);
      __syncthreads();

      if (global_queue_idx != -1) {
        if (threadIdx.x < blockDim.x - count)
          block_queue[tail + threadIdx.x] =
              globalQueue->buffer[global_queue_idx + threadIdx.x];
        __syncthreads();

        if (threadIdx.x == 0) {
          tail += blockDim.x - count;
          count = blockDim.x;
          globalQueue->ackEnqueue(blockDim.x - count);
        }
      }
    }

    if (buf_elems == 0) {
      if (ensureDequeue<BLOCK_QUEUE_SIZE>(&count)) {
        interact_buf[0] = block_queue[atomicAdd(&head, 1) % BLOCK_QUEUE_SIZE];
      } else
        continue;
    } else {
      buf_elems--;
    }

    bool switch_nodes = interact_buf[buf_elems].n1->header.kind >
                        interact_buf[0].n2->header.kind;
    // If there is enough register space, consider loading into register
    NodeElement *left =
        switch_nodes ? interact_buf[buf_elems].n2 : interact_buf[buf_elems].n1;
    NodeElement *right =
        switch_nodes ? interact_buf[buf_elems].n1 : interact_buf[buf_elems].n2;

    // Load actions
    Action *actions = actions_map +
                      actMapIndex(left->header.kind, right->header.kind) +
                      MAX_ACTIONS * (left->header.value == right->header.value);
    uint8_t next_action = 0;

    NodeElement *active_pair[2] = {left, right};

    NodeElement *new_nodes[MAX_NEW_NODES];
    uint8_t next_new = 0;

    // TODO: Test doing it all in a single loop
    while (next_action < MAX_ACTIONS && actions[next_action].kind == NEW) {
      NewNodeAction nna = actions[next_action].action.new_node;
      uint32_t value;
      if (nna.value == -1)
        value = left->header.value;
      else if (nna.value == -2)
        value = right->header.value;
      else if (nna.value == -3)
        value = reinterpret_cast<std::uintptr_t>(left);
      else
        value = nna.value;

      new_nodes[next_new] = (NodeElement *)malloc(
          sizeof(NodeElement) * (1 + 2 * (NODE_ARITIES[nna.kind] + 1)));
      // Should do this in one memory operation
      new_nodes[next_new][0] = {{nna.kind, value}};

      next_action++;
      next_new++;
    }

    // Perform connect actions
    while (next_action < MAX_ACTIONS && actions[next_action].kind == CONNECT) {
      ConnectAction ca = actions[next_action].action.connect;

      NodeElement *n1, *n2;
      uint64_t p1 = connect_p(ca.c1), p2 = connect_p(ca.c2);

      if (connect_g(ca.c1) == ACTIVE_PAIR) {
        n1 = active_pair[connect_n(ca.c1)];
      } else if (connect_g(ca.c1) == VARS) {
        n1 = active_pair[connect_n(ca.c1)][1 + 2 * (connect_p(ca.c1) + 1)]
                 .port_node;
        p1 = active_pair[connect_n(ca.c1)][1 + 2 * (connect_p(ca.c1) + 1)]
                 .port_port;
      } else {
        n1 = new_nodes[connect_n(ca.c1)];
      }

      if (connect_g(ca.c2) == ACTIVE_PAIR) {
        n2 = active_pair[connect_n(ca.c2)];
      } else if (connect_g(ca.c2) == VARS) {
        n2 = active_pair[connect_n(ca.c2)][1 + 2 * (connect_p(ca.c2) + 1)]
                 .port_node;
        p2 = active_pair[connect_n(ca.c2)][1 + 2 * (connect_p(ca.c2) + 1)]
                 .port_port;
      } else {
        n2 = new_nodes[connect_n(ca.c2)];
      }

      // Potential contention
      if (connect_g(ca.c1) == VARS) {
        uint64_t old_node =
            reinterpret_cast<uintptr_t>(active_pair[connect_n(ca.c1)]);
        uint64_t old_port = connect_p(ca.c1) + 1;
        unsigned long long assumed_node, assumed_port;
        do {
          assumed_node = old_node;
          assumed_port = old_port;
          old_node = atomicCAS((unsigned long long *)n1 + 1 + 2 * p1,
                               assumed_node, reinterpret_cast<uintptr_t>(n2));
          // Chance of failure here!
          old_port = atomicCAS((unsigned long long *)n1 + 1 + 2 * p1 + 1,
                               assumed_port, p2);
        } while (assumed_node != old_node || assumed_port != old_port);
      } else {
        // We want these assignments to be a single memory write
        ((Port *)(n1 + 1))[p1] = {(NodeElement *)n2, p2};
      }

      if (connect_g(ca.c2) == VARS) {
        uint64_t old_node =
            reinterpret_cast<uintptr_t>(active_pair[connect_n(ca.c2)]);
        uint64_t old_port = connect_p(ca.c2) + 1;
        unsigned long long assumed_node, assumed_port;
        do {
          assumed_node = old_node;
          assumed_port = old_port;
          old_node = atomicCAS((unsigned long long *)n2 + 1 + 2 * p2,
                               assumed_node, reinterpret_cast<uintptr_t>(n1));
          // Chance of failure here!
          old_port = atomicCAS((unsigned long long *)n2 + 1 + 2 * p2 + 1,
                               assumed_port, p1);
        } while (assumed_node != old_node || assumed_port != old_port);
      } else {
        // We want these assignments to be a single memory write
        ((Port *)(n2 + 1))[p2] = {(NodeElement *)n1, p1};
      }

      // Add any new interactions
      if (p1 == 0 && p2 == 0) {
        if (buf_elems < 5) {
          interact_buf[buf_elems] = {n1, n2};
          buf_elems++;
        } else {
          // WARNING: awful code!
          // If block queue full, enqueue onto global queue
          while (!ensureEnqueue<BLOCK_QUEUE_SIZE>(&count)) {
            int64_t g_q_idx = -1;
            while (g_q_idx != -1) {
              globalQueue->enqueue(&g_q_idx, 1);
            }
            globalQueue->buffer[g_q_idx] = {n1, n2};
            globalQueue->ackEnqueue(1);
          }
          block_queue[atomicAdd(&tail, 1) % BLOCK_QUEUE_SIZE] = {n1, n2};
        }
      }

      next_action++;
    }

    // Perform Free actions
    while (next_action < MAX_ACTIONS && actions[next_action].kind == FREE) {
      if (actions[next_action].action.free) {
        free(left);
      } else {
        free(right);
      }

      next_action++;
    }
  }

  // at the end copy network back to init_network_d
  // final network must be smaller than init network so there will be space
}
