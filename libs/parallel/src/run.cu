#include "../include/parallel/actions.cuh"
#include "../include/parallel/inet.cuh"
#include "../include/parallel/network.cuh"
#include "../include/parallel/queue.cuh"
#include "generate/grammar.hpp"
#include <cstdlib>
#include <memory>
#include <string>

void checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    const char *errorStr = hipGetErrorString(err);
    fprintf(stderr,
            "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
            "line %i.\n",
            err, errorStr, file, line);
    exit(EXIT_FAILURE);
  }
}

void checkCudaErrors(hipError_t err) {
  checkCudaErrors(err, __FILE__, __LINE__);
}

void run(std::unique_ptr<grammar::Grammar> grammar, std::string input_string) {
  hipDeviceProp_t *prop;
  checkCudaErrors(hipGetDeviceProperties(prop, 0));

  dim3 grid_dims(16, 1, 1);
  dim3 block_dims(256, 1, 1);

  initActions();

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(NODE_ARITIES), NODE_ARITIES_H,
                                     sizeof(uint8_t) * NODE_KINDS));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(actions_map), actions_map_h,
                                     sizeof(Action) * ACTIONS_MAP_SIZE));

  std::vector<grammar::Token> tokens = grammar->stringToTokens(input_string);
  HostINetwork host_network(grammar->getStackActions(), tokens);

  size_t interactions_size =
      sizeof(Interaction) * host_network.getInteractions();
  size_t network_size = sizeof(NodeElement) * host_network.getNetworkSize();

  Interaction *interactions = (Interaction *)malloc(interactions_size);
  NodeElement *network_h = (NodeElement *)malloc(network_size);

  host_network.initNetwork(network_h, interactions);

  // Initialize global queue such that the first set of interactions can be
  // immediately loaded by the threads
  InteractionQueue<MAX_INTERACTIONS_SIZE> globalQueue_h(
      interactions, interactions_size, grid_dims.x * block_dims.x);
  InteractionQueue<MAX_INTERACTIONS_SIZE> *globalQueue_d;
  checkCudaErrors(hipMalloc((void **)&globalQueue_d,
                             sizeof(InteractionQueue<MAX_INTERACTIONS_SIZE>)));
  checkCudaErrors(hipMemcpy(globalQueue_d, &globalQueue_h,
                             sizeof(InteractionQueue<MAX_INTERACTIONS_SIZE>),
                             hipMemcpyHostToDevice));

  NodeElement *network_d;

  checkCudaErrors(hipMalloc((void **)&network_d, network_size));
  checkCudaErrors(
      hipMemcpy(network_d, network_h, network_size, hipMemcpyHostToDevice));

  hipDeviceSetLimit(hipLimitMallocHeapSize,
                     MAX_INTERACTIONS_SIZE * sizeof(Interaction) +
                         MAX_NETWORK_SIZE * sizeof(NodeElement));

  bool *global_done_d;
  bool global_done_h[grid_dims.x];

  // Invoke kernel
  runINet<<<grid_dims, block_dims>>>(network_d, globalQueue_d,
                                     interactions_size, global_done_d);

  checkCudaErrors(
      hipMemcpy(network_h, network_d, network_size, hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(globalQueue_d));
  checkCudaErrors(hipFree(network_d));

  // traverse network_h and retrieve parse
}
